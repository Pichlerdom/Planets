#include "planets.h"
#include "quadtree.h"

void move_planets(PlanetsArr* container, PConfig *pconfig, GPU_Mem *gpu_mem){

	Planet *h_planets = container->planets;

	int numberOfPlanets = container->size_arr;
	dim3 block(PLANET_BLOCK_N,PLANET_BLOCK_N);	
	dim3 grid(container->size_arr/PLANET_BLOCK_N,container->size_arr/PLANET_BLOCK_N);
	
	realloc_cuda(container, gpu_mem);

pthread_mutex_lock(&(container->planetsMutex));
	
	hipDeviceSynchronize();
		hipMemcpy((void*)gpu_mem->d_planets,
				   (void *)h_planets,
					gpu_mem->size_arr * sizeof(Planet),
				   hipMemcpyHostToDevice);
pthread_mutex_unlock(&(container->planetsMutex));
	
	calculate_f_sum_reduction<<<grid,block>>>(gpu_mem->d_planets,gpu_mem->d_f);

	calculate_f_sum<<<grid.x,block.x>>>(gpu_mem->d_planets,gpu_mem->d_f);

	move_planets_kernel<<<grid.y, block>>>(gpu_mem->d_planets);
	
	pthread_mutex_lock(&(container->planetsMutex));
			
	hipMemcpy(	(void *)h_planets,
				(void *)gpu_mem->d_planets,
				numberOfPlanets * sizeof(Planet),
				hipMemcpyDeviceToHost);
	pthread_mutex_unlock(&(container->planetsMutex));
		
}

void realloc_cuda(PlanetsArr* container, GPU_Mem *gpu_mem){
	if(container->size_arr != gpu_mem->size_arr){	

		hipFree((void **)(gpu_mem->d_f));
		hipFree((void **)(gpu_mem->d_planets));
		gpu_mem->size_arr = container->size_arr;
		if(hipSuccess != hipMalloc((void **)&(gpu_mem->d_planets),
									  gpu_mem->size_arr * sizeof(Planet))){
			printf("Planets memory allocation error!\n");
			return;
		}
		if(hipSuccess != hipMalloc((void **)&(gpu_mem->d_f),
									 gpu_mem->size_arr * PLANET_BLOCK_DIM * sizeof(Vec))){	
			printf("Force memory allocation error!\n"); 
			return;
		}

	}
}

void remove_dead_planets(PlanetsArr* container){
	int c = container->size_arr;
	for(int i = 0; i < c ;i++){
		if(container->planets[i].mass < 0.0 ){
			for(int j = container->size_arr - 1; j >= i; j--){
				if(container->planets[j].mass > 0.0){
					container->planets[i] = container->planets[j];
					container->planets[j].mass = -1.0;
					break;	
				}
			}
			
		}
	}
	container->number = 0;
	while(container->planets[container->number].mass >= 0.0 && container->number < container->size_arr){
		container->number ++;
	}


	int delta  =container->size_arr-1 - container->number;
	if(delta >= PLANET_BLOCK_N){ 
		container->size_arr = container->size_arr - (delta -delta%PLANET_BLOCK_N);		
		container->planets = (Planet*) realloc(container->planets,container->size_arr * sizeof(Planet));

	}
}



void *main_calc_loop(void *arguments){
	//Display contains all the structures we need for rendering stuff
	ThreadArgs *args = (ThreadArgs*) arguments;
	PlanetsArr *container = args->container;
	PConfig *pconfig = args->pconfig;

	pthread_t qtree_thread;	
	pthread_create(&qtree_thread,NULL,&main_qtree_loop,arguments);
	
	uint32_t *calctime = &(args->calctime);

	uint32_t currTime = SDL_GetTicks();
	uint32_t frameTime = 0u;
	GPU_Mem gpu_mem;
	gpu_mem.size_arr = 0;


	while(!container->quit){
		
		currTime = SDL_GetTicks();
		
		move_planets(container, pconfig, &gpu_mem);

		pthread_mutex_lock(&(container->planetsMutex));
		
		remove_dead_planets(container);
		
		pthread_mutex_unlock(&(container->planetsMutex));


		//FPS stuff
		frameTime = SDL_GetTicks() - currTime;
		
		*calctime = frameTime;
		if(frameTime > MS_PER_TICK){
			frameTime = MS_PER_TICK;
		}
		SDL_Delay(MS_PER_TICK-frameTime);
	}
	
	printf("Calc thread exited:\n");
	return NULL;
}


void *main_qtree_loop(void *arguments){

	ThreadArgs *args = (ThreadArgs*) arguments;
	PlanetsArr *container = args->container;

	args->qtree = init_qtree(__QTREE_SIZE);
	QTree *qtree = args->qtree;
		
	uint32_t currTime = SDL_GetTicks();
	uint32_t frameTime = 0u;
	
	pthread_mutex_lock(&(args->qtreeMutex));

	while(!container->quit){
		currTime = SDL_GetTicks();

		pthread_mutex_lock(&(container->planetsMutex));

			
		construct_qtree(qtree, container->planets, container->number);

		pthread_mutex_unlock(&(args->qtreeMutex));

		pthread_mutex_unlock(&(container->planetsMutex));
	
		SDL_Delay(1);
		pthread_mutex_lock(&(args->qtreeMutex));
		args->qtree = qtree;
		clear_qtree(qtree);
	
		frameTime = SDL_GetTicks() - currTime;
		
		args->qtree_time = frameTime;
	}

}


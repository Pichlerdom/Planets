#include "planets.h"
#include "quadtree.h"

void move_planets(PlanetsArr* container, PConfig *pconfig, GPU_Mem *gpu_mem){
	Planet *h_planets = container->planets;
	
	int numberOfPlanets = container->size_arr;
	dim3 block(PLANET_BLOCK_N,PLANET_BLOCK_N);	
	dim3 grid(container->size_arr/PLANET_BLOCK_N,container->size_arr/PLANET_BLOCK_N);

	realloc_cuda(container, gpu_mem);

	hipDeviceSynchronize();
		hipMemcpy((void*)gpu_mem->d_planets,
				   (void *)h_planets,
					gpu_mem->size_arr * sizeof(Planet),
				   hipMemcpyHostToDevice);

	calculate_f_sum_reduction<<<grid,block>>>(gpu_mem->d_planets,gpu_mem->d_f);

	calculate_f_sum<<<grid.x,block.x>>>(gpu_mem->d_planets,gpu_mem->d_f);

	move_planets_kernel<<<grid.y, block>>>(gpu_mem->d_planets);
	
			
	hipMemcpy(	(void *)h_planets,
				(void *)gpu_mem->d_planets,
				numberOfPlanets * sizeof(Planet),
				hipMemcpyDeviceToHost);
		
}

void realloc_cuda(PlanetsArr* container, GPU_Mem *gpu_mem){
	if(container->size_arr != gpu_mem->size_arr){	

		hipFree((void **)(gpu_mem->d_f));
		hipFree((void **)(gpu_mem->d_planets));


		gpu_mem->size_arr = container->size_arr;
		if(hipSuccess != hipMalloc((void **)&(gpu_mem->d_planets),
									  gpu_mem->size_arr * sizeof(Planet))){
			printf("Planets memory allocation error!\n");
			return;
		}
		if(hipSuccess != hipMalloc((void **)&(gpu_mem->d_f),
									 gpu_mem->size_arr * PLANET_BLOCK_DIM * sizeof(Vec))){	
			printf("Force memory allocation error!\n"); 
			return;
		}

	}
}

void remove_dead_planets(PlanetsArr* container){
	int c = container->size_arr;
	for(int i = 0; i < c ;i++){
		if(container->planets[i].mass < 0.0 ){
			for(int j = container->size_arr - 1; j >= i; j--){
				if(container->planets[j].mass > 0.0){
					container->planets[i] = container->planets[j];
					container->planets[j].mass = -1.0;
					container->planets[j].r = 0;
					break;	
				}
			}
			
		}
	}
	container->number = 0;
	while(container->planets[container->number].mass >= 0.0 && container->number < container->size_arr){
		container->number ++;
	}


	int delta  =container->size_arr-1 - container->number;
	if(delta >= PLANET_BLOCK_N){ 
		container->size_arr = container->size_arr - (delta -delta%PLANET_BLOCK_N);		
		container->planets = (Planet*) realloc(container->planets,container->size_arr * sizeof(Planet));

	}
}



void *main_calc_loop(void *arguments){
	//Display contains all the structures we need for rendering stuff
	ThreadArgs *args = (ThreadArgs*) arguments;
	PlanetsArr *container = args->container;
	PConfig *pconfig = args->pconfig;

	pthread_t qtree_thread;	
	pthread_create(&qtree_thread,NULL,&main_qtree_loop,arguments);
	
	uint32_t *calctime = &(args->calctime);

	uint32_t currTime = SDL_GetTicks();
	uint32_t frameTime = 0u;
	GPU_Mem gpu_mem;
	gpu_mem.size_arr = 0;


	while(!container->quit){
			currTime = SDL_GetTicks();
	

		
	pthread_mutex_lock(&(container->planetsMutex));

		move_planets(container, pconfig, &gpu_mem);
		container->modified = true;
	pthread_mutex_unlock(&(container->planetsMutex));

		//FPS stuff
		frameTime = SDL_GetTicks() - currTime;
		
		*calctime = frameTime;
		if(frameTime > MS_PER_TICK){
			frameTime = MS_PER_TICK-1;
		}
		SDL_Delay(MS_PER_TICK-frameTime);
	}
	
	printf("Calc thread exited:\n");
	return NULL;
}


void *main_qtree_loop(void *arguments){

	ThreadArgs *args = (ThreadArgs*) arguments;
	PlanetsArr *container = args->container;

	args->qtree = init_qtree(__QTREE_SIZE);
	QTree *qtree = args->qtree;
		
	uint32_t currTime = SDL_GetTicks();
	uint32_t frameTime = 0u;

	while(!container->quit){

	pthread_mutex_lock(&(args->qtreeMutex));
		

		clear_qtree(qtree);
		qtree = init_qtree(__QTREE_SIZE);
			
		args->qtree = qtree;

	pthread_mutex_lock(&(container->planetsMutex));

		currTime = SDL_GetTicks();

		construct_qtree(qtree, container->planets,container->number);
		
		remove_dead_planets(container);

		pthread_mutex_unlock(&(container->planetsMutex));
	

	pthread_mutex_unlock(&(args->qtreeMutex));

		frameTime = SDL_GetTicks() - currTime;
		
		args->qtree_time = frameTime;
		if(frameTime > MS_PER_TICK){
			frameTime = MS_PER_TICK-1;
		}
		SDL_Delay(MS_PER_TICK-frameTime);
	}
	return 0;
}

